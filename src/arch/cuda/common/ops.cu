#include "ops.cuh"

namespace orthrus::models::common::cuda {

void CHECK_CUBLAS( const hipblasStatus_t err, const std::source_location location )
{
  if ( err != HIPBLAS_STATUS_SUCCESS ) {
    LOG( FATAL ) << "CUBLAS error " << cublasGetStatusName( err ) << ": " << cublasGetStatusString( err ) << " ("
                 << location.file_name() << ":" << std::to_string( location.line() ) << ")";
  }
}

void CHECK_CUDA( const hipError_t err, const std::source_location location )
{
  if ( err != hipSuccess ) {
    LOG( FATAL ) << "CUDA error " << hipGetErrorName( err ) << ": " << hipGetErrorString( err ) << " ("
                 << location.file_name() << ":" << std::to_string( location.line() ) << ")";
  }
}

}
